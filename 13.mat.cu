#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define N 2000 // Matrix size (NxN)

// CUDA kernel for matrix multiplication
__global__ void matrix_multiply(float *A, float *B, float *C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        float sum = 0.0f;
        for (int k = 0; k < n; k++) {
            sum += A[row * n + k] * B[k * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main() {
    int size = N * N * sizeof(float);
    float *A, *B, *C;
    float *d_A, *d_B, *d_C;

    // Allocate host memory
    A = (float *)malloc(size);
    B = (float *)malloc(size);
    C = (float *)malloc(size);

    // Initialize matrices
    for (int i = 0; i < N * N; i++) {
        A[i] = 1; // Example values
        B[i] = 2;
    }

    // Allocate device memory
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // Copy data to device
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 threads_per_block(16, 16);
    dim3 blocks_per_grid((N + threads_per_block.x - 1) / threads_per_block.x,
                         (N + threads_per_block.y - 1) / threads_per_block.y);

    // Launch kernel
    matrix_multiply<<<blocks_per_grid, threads_per_block>>>(d_A, d_B, d_C, N);

    // Copy result back to host
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Print one result element (e.g., C[0][0])
    printf("C[0][0] = %f\n", C[0]);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(A);
    free(B);
    free(C);

    return 0;
}
